
#include "eval_density.cuh"
#include "basis_to_gpu.cuh"
#include "cuda_utils.cuh"
#include "run.cuh"

using namespace  chemtools;

__host__ void chemtools::evaluate_scalar_quantity_density(
    const MolecularBasis& basis,
          bool            do_segmented_basis,
    const bool            disp,
          double*         d_output_iter,
    const double*         d_points_iter,
    const int             knumb_points_iter,
    const int             k_total_numb_contractions,
          dim3            threadsPerBlock,
          dim3            grid,
      hipFuncCache_t l1_over_shared
) {
  // Set the temprory kernel to prefer L1 cache over shared memory.
  chemtools::cuda_check_errors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(chemtools::eval_AOs_from_constant_memory_on_any_grid),
                                                      l1_over_shared));

  // Start at the first shell
  std::size_t i_shell = 0;                              // Controls which shells are in constant memory
  std::size_t numb_basis_funcs_completed = 0;           // Controls where to update the next set of contractions
  std::array<std::size_t, 2> i_shell_and_numb_basis{};  // Used to update how many shells and contractions are in
  //    constant memory.
  // Increment through each shell that can fit inside constant memory.
  while (i_shell < basis.numb_contracted_shells()) {
    // Transfer the basis-set to constant memory
    i_shell_and_numb_basis = chemtools::add_mol_basis_to_constant_memory_array(
        basis, do_segmented_basis, disp, i_shell
    );

    // Evaluate the function over the GPU
    chemtools::eval_AOs_from_constant_memory_on_any_grid<<<grid, threadsPerBlock>>>(
        d_output_iter, d_points_iter, knumb_points_iter, k_total_numb_contractions,
            static_cast<int>(numb_basis_funcs_completed));
    hipDeviceSynchronize();

    // Update to the next begining of the shells and contractions
    i_shell = i_shell_and_numb_basis[0];
    numb_basis_funcs_completed += i_shell_and_numb_basis[1];
  }
}