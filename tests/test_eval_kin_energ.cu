#include "hip/hip_runtime.h"
#include "catch.hpp"

#include <random>
#include <algorithm>
#include <iterator>

#include <pybind11/embed.h>
#include <pybind11/numpy.h>

#include "../include/iodata.h"
#include "../include/utils.h"
#include "../include/cuda_utils.cuh"
#include "../include/basis_to_gpu.cuh"
#include "../include/eval_kin_energ.cuh"

namespace py = pybind11;
using namespace py::literals;


TEST_CASE( "Test Positive Definite Kinetic Energy Density Against gbasis", "[evaluate_posdef_kinetic_energy]" ) {
  //py::initialize_interpreter();  // Open up the python interpretor for this test.
  {  // Need this so that the python object doesn't outline the interpretor.
    // Get the IOdata object from the fchk file.
    std::string fchk_file = GENERATE(
        "./tests/data/atom_01_H_N01_M2_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/atom_he.fchk",
        "./tests/data/atom_be.fchk",
        "./tests/data/atom_be_f_pure_orbital.fchk",
        "./tests/data/atom_be_f_cartesian_orbital.fchk",
        "./tests/data/atom_kr.fchk",
        "./tests/data/atom_o.fchk",
        "./tests/data/atom_c_g_pure_orbital.fchk",
        "./tests/data/atom_mg.fchk",
        "./tests/data/E948_rwB97XD_def2SVP.fchk",
        "./tests/data/test.fchk",
        "./tests/data/test2.fchk",
        "./tests/data/atom_08_O_N08_M3_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/atom_08_O_N09_M2_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/h2o.fchk",
        "./tests/data/ch4.fchk",
        "./tests/data/qm9_000092_HF_cc-pVDZ.fchk",
        "./tests/data/qm9_000104_PBE1PBE_pcS-3.fchk"
    );
    std::cout << "Pos Def KED FCHK FILE %s \n" << fchk_file << std::endl;
    chemtools::IOData iodata = chemtools::get_molecular_basis_from_fchk(fchk_file);

    // Gemerate random grid.
    int numb_pts = 750000;
    std::vector<double> points(3 * numb_pts);
    std::random_device rnd_device;
    std::mt19937  merseene_engine {rnd_device()};
    std::uniform_real_distribution<double> dist {-5, 5};
    auto gen = [&dist, &merseene_engine](){return dist(merseene_engine);};
    std::generate(points.begin(), points.end(), gen);

    // Calculate Kinetic Energy
    std::vector<double> kinetic_dens_result = chemtools::evaluate_positive_definite_kinetic_density(
        iodata, points.data(), numb_pts
        );

    // COnvert them (with copy) to python objects so that they can be transfered.
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
        py_result = chemtools::as_pyarray_from_vector(kinetic_dens_result);
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
        py_points = chemtools::as_pyarray_from_vector(points);

    const int nbasis = iodata.GetOrbitalBasis().numb_basis_functions();

    auto locals = py::dict("points"_a = py_points,
                           "true_result"_a = py_result,
                           "fchk_path"_a = fchk_file,
                           "numb_pts"_a = numb_pts,
                           "nbasis"_a = nbasis);
    py::exec(R"(
import numpy as np
from gbasis.evals.density import evaluate_posdef_kinetic_energy_density
from iodata import load_one
from gbasis.wrappers import from_iodata

iodata = load_one(fchk_path)
basis = from_iodata(iodata)
rdm = (iodata.mo.coeffs * iodata.mo.occs).dot(iodata.mo.coeffs.T)
points = points.reshape((numb_pts, 3), order="F")
points = np.array(points, dtype=np.float64)

indices_to_compute = np.random.choice(np.arange(len(points)), size=10000)
true_result = true_result[indices_to_compute]
points = points[indices_to_compute, :]

kin_dens = evaluate_posdef_kinetic_energy_density(rdm, basis, points)
err = np.abs(kin_dens - true_result)
result = np.all(err < 1e-8)
print("Kinetic Density Mean, Max, STD Error ", np.mean(err), np.max(err), np.std(err))
assert result, "Kinetic Energy of Electron Density on GPU doesn't match gbasis."
    )", py::globals(), locals);
  } // Need this so that the python object doesn't outline the interpretor when we close it up.
}


TEST_CASE( "Test General Kinetic Energy Density Against gbasis", "[evaluate_general_kinetic_energy]" ) {
  //py::initialize_interpreter();  // Open up the python interpretor for this test.
  {  // Need this so that the python object doesn't outline the interpretor.
    // Get the IOdata object from the fchk file.
    std::string fchk_file = GENERATE(
        "./tests/data/atom_01_H_N01_M2_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/atom_he.fchk",
        "./tests/data/atom_be.fchk",
        "./tests/data/atom_be_f_pure_orbital.fchk",
        "./tests/data/atom_be_f_cartesian_orbital.fchk",
        "./tests/data/atom_kr.fchk",
        "./tests/data/atom_o.fchk",
        "./tests/data/atom_c_g_pure_orbital.fchk",
        "./tests/data/atom_mg.fchk",
        "./tests/data/E948_rwB97XD_def2SVP.fchk",
        "./tests/data/test.fchk",
        "./tests/data/test2.fchk",
        "./tests/data/atom_08_O_N08_M3_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/atom_08_O_N09_M2_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/h2o.fchk",
        "./tests/data/ch4.fchk",
        "./tests/data/qm9_000092_HF_cc-pVDZ.fchk",
        "./tests/data/qm9_000104_PBE1PBE_pcS-3.fchk"
    );
    std::cout << "General KED FCHK FILE %s \n" << fchk_file << std::endl;
    chemtools::IOData iodata = chemtools::get_molecular_basis_from_fchk(fchk_file);

    // Gemerate random grid.
    int numb_pts = 10000;
    std::vector<double> points(3 * numb_pts);
    std::random_device rnd_device;
    std::mt19937  merseene_engine {rnd_device()};
    std::uniform_real_distribution<double> dist {-5, 5};
    auto gen = [&dist, &merseene_engine](){return dist(merseene_engine);};
    std::generate(points.begin(), points.end(), gen);

    // Calculate General Kinetic Energy at alpha = 0.5
    std::uniform_real_distribution<double> alpha_gen {-5, 5};
    double alpha = alpha_gen(merseene_engine);
    std::vector<double> laplacian_result = chemtools::evaluate_general_kinetic_energy_density(
        iodata, alpha, points.data(), numb_pts
    );

    // COnvert them (with copy) to python objects so that they can be transfered.
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
        py_result = chemtools::as_pyarray_from_vector(laplacian_result);
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
        py_points = chemtools::as_pyarray_from_vector(points);

    const int nbasis = iodata.GetOrbitalBasis().numb_basis_functions();

    auto locals = py::dict("points"_a = py_points,
                           "true_result"_a = py_result,
                           "fchk_path"_a = fchk_file,
                           "numb_pts"_a = numb_pts,
                           "nbasis"_a = nbasis,
                           "alpha"_a = alpha);
    py::exec(R"(
import numpy as np
from gbasis.evals.density import evaluate_general_kinetic_energy_density
from iodata import load_one
from gbasis.wrappers import from_iodata

iodata = load_one(fchk_path)
basis = from_iodata(iodata)
rdm = (iodata.mo.coeffs * iodata.mo.occs).dot(iodata.mo.coeffs.T)
points = points.reshape((numb_pts, 3), order="F")
points = np.array(points, dtype=np.float64)

kin_dens = evaluate_general_kinetic_energy_density(rdm, basis, points, alpha=alpha)
err = np.abs(kin_dens - true_result)
result = np.all(err < 1e-8)
print("General Kinetic Density Mean, Max, STD Error ", np.mean(err), np.max(err), np.std(err))
assert result, "General Kinetic Energy of Electron Density on GPU doesn't match gbasis."
    )", py::globals(), locals);
  } // Need this so that the python object doesn't outline the interpretor when we close it up.
}