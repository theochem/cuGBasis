#include "hip/hip_runtime.h"
#include "catch.hpp"

#include <pybind11/embed.h>
#include <pybind11/numpy.h>

#include "../include/iodata.h"
#include "../include/eval_mo.cuh"
#include "../include/cuda_utils.cuh"
#include "../include/basis_to_gpu.cuh"
#include "../include/utils.h"

namespace py = pybind11;
using namespace py::literals;


TEST_CASE( "Test Molecular Orbitals Against gbasis on random grid", "[evaluate_mol_orbitals_on_any_grid]" ) {
  {  // Need this so that the python object doesn't outline the interpretor.
    // Evaluate the electron density of this example.
    std::string fchk_file = GENERATE(
        "./tests/data/atom_01_H_N01_M2_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/atom_he.fchk",
        "./tests/data/atom_be.fchk",
        "./tests/data/atom_be_f_pure_orbital.fchk",
        "./tests/data/atom_be_f_cartesian_orbital.fchk",
        "./tests/data/atom_kr.fchk",
        "./tests/data/atom_o.fchk",
        "./tests/data/atom_c_g_pure_orbital.fchk",
        "./tests/data/atom_mg.fchk",
        "./tests/data/E948_rwB97XD_def2SVP.fchk",
        "./tests/data/test.fchk",
        "./tests/data/test2.fchk",
        "./tests/data/atom_08_O_N08_M3_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/atom_08_O_N09_M2_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/h2o.fchk",
        "./tests/data/ch4.fchk",
        "./tests/data/qm9_000092_HF_cc-pVDZ.fchk",
        "./tests/data/qm9_000104_PBE1PBE_pcS-3.fchk",
        "./tests/data/DUTLAF10_0_q000_m01_k00_force_uwb97xd_def2svpd.fchk"
    );
    printf("Test: %s \n", fchk_file.c_str());
    chemtools::IOData iodata = chemtools::get_molecular_basis_from_fchk(fchk_file);

    // Gemerate random grid.
    int numb_pts = 1000;
    std::vector<double> points(3 * numb_pts);
    std::random_device rnd_device;
    std::mt19937  merseene_engine {rnd_device()};
    std::uniform_real_distribution<double> dist {-5, 5};
    auto gen = [&dist, &merseene_engine](){return dist(merseene_engine);};
    std::generate(points.begin(), points.end(), gen);

    // Evaluate electron density on the cube
    printf("Evaluate Molecular Orbitals \n");
    std::vector<double> result = chemtools::eval_MOs(iodata, points.data(), numb_pts);

    //Transfer result to pybind11 without copying
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast> py_result =
        chemtools::as_pyarray_from_vector(result);
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
        py_points = chemtools::as_pyarray_from_vector(points);

    auto locals = py::dict(
        "true_result"_a=py_result, "fchk_path"_a=fchk_file, "points"_a = py_points, "numb_pts"_a = numb_pts,
        "nbasis"_a = iodata.GetOneRdmShape()
    );
    py::exec(R"(
import numpy as np
from gbasis.evals.density import evaluate_basis
from iodata import load_one
from gbasis.wrappers import from_iodata
try:
    from iodata.convert import convert_conventions, HORTON2_CONVENTIONS
except (ImportError, ModuleNotFoundError):
    from iodata.basis import convert_conventions, HORTON2_CONVENTIONS
true_result = true_result.reshape((nbasis, numb_pts), order="F")

iodata = load_one(fchk_path)
basis = from_iodata(iodata)
coeffs = iodata.mo.coeffsa
rdm = coeffs.T

points = points.reshape((numb_pts, 3), order="F")
points = np.array(points, dtype=np.float64)

random_indices = np.unique(np.random.randint(0, len(points), 10000))
points = points[random_indices, :]

mol_orbitals = rdm.dot(evaluate_basis(basis, points))
err = np.abs(mol_orbitals - true_result[:, random_indices])
result = np.all(err < 1e-8)
print(f"Max Error {np.max(err)}     Mean Err {np.mean(err)}    Std Err {np.std(err)}")
assert result, "Molecular orbitals on GPU doesn't match gbasis."
    )", py::globals(), locals);

    if (!locals["result"].cast<bool>()) {
      REQUIRE(true);
    }
  } // Need this so that the python object doesn't outline the interpretor when we close it up.
}



TEST_CASE( "Test Molecular Orbitals Derivatives Against gbasis on random grid", "[evaluate_mol_orbitals_deriv_on_any_grid]" ) {
    {  // Need this so that the python object doesn't outline the interpretor.
        // Evaluate the electron density of this example.
        std::string fchk_file = GENERATE(
            "./tests/data/atom_01_H_N01_M2_ub3lyp_ccpvtz_g09.fchk",
            "./tests/data/atom_he.fchk",
            "./tests/data/atom_be.fchk",
            "./tests/data/atom_be_f_pure_orbital.fchk",
            "./tests/data/atom_be_f_cartesian_orbital.fchk",
            "./tests/data/atom_kr.fchk",
            "./tests/data/atom_o.fchk",
            "./tests/data/atom_c_g_pure_orbital.fchk",
            "./tests/data/atom_mg.fchk",
            "./tests/data/E948_rwB97XD_def2SVP.fchk",
            "./tests/data/test.fchk",
            "./tests/data/test2.fchk",
            "./tests/data/atom_08_O_N08_M3_ub3lyp_ccpvtz_g09.fchk",
            "./tests/data/atom_08_O_N09_M2_ub3lyp_ccpvtz_g09.fchk",
            "./tests/data/h2o.fchk",
            "./tests/data/ch4.fchk",
            "./tests/data/qm9_000092_HF_cc-pVDZ.fchk",
            "./tests/data/qm9_000104_PBE1PBE_pcS-3.fchk",
            "./tests/data/DUTLAF10_0_q000_m01_k00_force_uwb97xd_def2svpd.fchk"
        );
        printf("Test: %s \n", fchk_file.c_str());
        chemtools::IOData iodata = chemtools::get_molecular_basis_from_fchk(fchk_file);
        
        // Gemerate random grid.
        int numb_pts = 1000;
        std::vector<double> points(3 * numb_pts);
        std::random_device rnd_device;
        std::mt19937  merseene_engine {rnd_device()};
        std::uniform_real_distribution<double> dist {-5, 5};
        auto gen = [&dist, &merseene_engine](){return dist(merseene_engine);};
        std::generate(points.begin(), points.end(), gen);
        
        // Evaluate electron density on the cube
        printf("Evaluate First Derivative Molecular Orbitals \n");
        std::vector<double> result = chemtools::eval_MOs_derivs(iodata, points.data(), numb_pts);
        
        //Transfer result to pybind11 without copying
        pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast> py_result =
                                      chemtools::as_pyarray_from_vector(result);
        pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
            py_points = chemtools::as_pyarray_from_vector(points);
        
        auto locals = py::dict(
            "true_result"_a=py_result, "fchk_path"_a=fchk_file, "points"_a = py_points, "numb_pts"_a = numb_pts,
            "nbasis"_a = iodata.GetOneRdmShape()
        );
        py::exec(R"(
import numpy as np
from gbasis.evals.eval_deriv import evaluate_deriv_basis
from iodata import load_one
from gbasis.wrappers import from_iodata
try:
    from iodata.convert import convert_conventions, HORTON2_CONVENTIONS
except (ImportError, ModuleNotFoundError):
    from iodata.basis import convert_conventions, HORTON2_CONVENTIONS
true_result = true_result.reshape((3, numb_pts, nbasis), order="C")

iodata = load_one(fchk_path)
basis = from_iodata(iodata)
coeffs = iodata.mo.coeffs
rdm = (coeffs * iodata.mo.occs).dot(coeffs.T)

points = points.reshape((numb_pts, 3), order="F")
points = np.array(points, dtype=np.float64)

random_indices = np.unique(np.random.randint(0, len(points), 1000))
points = points[random_indices, :]

for i, ord in enumerate([[1, 0, 0], [0, 1, 0], [0, 0, 1]]):
    ord = np.array(ord)
    mat = evaluate_deriv_basis(basis, points, ord)
    mol_orbitals = rdm.dot(mat)

    err = np.abs(mol_orbitals - true_result[i, random_indices, :].T)
    result = np.all(err < 1e-8)
    print(f"Max Error {np.max(err)}     Mean Err {np.mean(err)}    Std Err {np.std(err)}")
    assert result, "Molecular orbitals on GPU doesn't match gbasis."
    )", py::globals(), locals);
        
        if (!locals["result"].cast<bool>()) {
            REQUIRE(true);
        }
    } // Need this so that the python object doesn't outline the interpretor when we close it up.
}


TEST_CASE( "Test Molecular Orbitals Sec Derivatives Against gbasis on random grid", "[evaluate_mol_orbitals_sec_deriv_on_any_grid]" ) {
    {  // Need this so that the python object doesn't outline the interpretor.
        // Evaluate the electron density of this example.
        std::string fchk_file = GENERATE(
            "./tests/data/atom_01_H_N01_M2_ub3lyp_ccpvtz_g09.fchk",
            "./tests/data/atom_he.fchk",
            "./tests/data/atom_be.fchk",
            "./tests/data/atom_be_f_pure_orbital.fchk",
            "./tests/data/atom_be_f_cartesian_orbital.fchk",
            "./tests/data/atom_kr.fchk",
            "./tests/data/atom_o.fchk",
            "./tests/data/atom_c_g_pure_orbital.fchk",
            "./tests/data/atom_mg.fchk",
            "./tests/data/E948_rwB97XD_def2SVP.fchk",
            "./tests/data/test.fchk",
            "./tests/data/test2.fchk",
            "./tests/data/atom_08_O_N08_M3_ub3lyp_ccpvtz_g09.fchk",
            "./tests/data/atom_08_O_N09_M2_ub3lyp_ccpvtz_g09.fchk",
            "./tests/data/h2o.fchk",
            "./tests/data/ch4.fchk",
            "./tests/data/qm9_000092_HF_cc-pVDZ.fchk",
            "./tests/data/qm9_000104_PBE1PBE_pcS-3.fchk",
            "./tests/data/DUTLAF10_0_q000_m01_k00_force_uwb97xd_def2svpd.fchk"
        );
        printf("Test: %s \n", fchk_file.c_str());
        chemtools::IOData iodata = chemtools::get_molecular_basis_from_fchk(fchk_file);
        
        // Gemerate random grid.
        int numb_pts = 1000;
        std::vector<double> points(3 * numb_pts);
        std::random_device rnd_device;
        std::mt19937  merseene_engine {rnd_device()};
        std::uniform_real_distribution<double> dist {-5, 5};
        auto gen = [&dist, &merseene_engine](){return dist(merseene_engine);};
        std::generate(points.begin(), points.end(), gen);
        
        // Evaluate electron density on the cube
        printf("Evaluate Second Derivatives Molecular Orbitals \n");
        std::vector<double> result = chemtools::eval_MOs_second_derivs(iodata, points.data(), numb_pts);
        
        //Transfer result to pybind11 without copying
        pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast> py_result =
                                      chemtools::as_pyarray_from_vector(result);
        pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
            py_points = chemtools::as_pyarray_from_vector(points);
        
        auto locals = py::dict(
            "true_result"_a=py_result, "fchk_path"_a=fchk_file, "points"_a = py_points, "numb_pts"_a = numb_pts,
            "nbasis"_a = iodata.GetOneRdmShape()
        );
        py::exec(R"(
import numpy as np
from gbasis.evals.eval_deriv import evaluate_deriv_basis
from iodata import load_one
from gbasis.wrappers import from_iodata
try:
    from iodata.convert import convert_conventions, HORTON2_CONVENTIONS
except (ImportError, ModuleNotFoundError):
    from iodata.basis import convert_conventions, HORTON2_CONVENTIONS
true_result = true_result.reshape((6, numb_pts, nbasis), order="C")

iodata = load_one(fchk_path)
basis = from_iodata(iodata)
coeffs = iodata.mo.coeffs
rdm = (coeffs * iodata.mo.occs).dot(coeffs.T)

points = points.reshape((numb_pts, 3), order="F")
points = np.array(points, dtype=np.float64)

random_indices = np.unique(np.random.randint(0, len(points), 1000))
points = points[random_indices, :]

for i, ord in enumerate([[2, 0, 0], [1, 1, 0], [1, 0, 1], [0, 2, 0], [0, 1, 1], [0, 0, 2]]):
    ord = np.array(ord)
    mat = evaluate_deriv_basis(basis, points, ord)
    mol_orbitals = rdm.dot(mat)

    err = np.abs(mol_orbitals - true_result[i, random_indices, :].T)
    result = np.all(err < 1e-8)
    print(f"Max Error {np.max(err)}     Mean Err {np.mean(err)}    Std Err {np.std(err)}")
    assert result, "Molecular orbitals on GPU doesn't match gbasis."
    )", py::globals(), locals);
        
        if (!locals["result"].cast<bool>()) {
            REQUIRE(true);
        }
    } // Need this so that the python object doesn't outline the interpretor when we close it up.
}