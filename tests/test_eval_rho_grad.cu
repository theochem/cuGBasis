#include "hip/hip_runtime.h"
#include "catch.hpp"

#include <random>
#include <algorithm>
#include <iterator>

#include <pybind11/embed.h>
#include <pybind11/stl_bind.h>
#include <pybind11/numpy.h>

#include "../include/iodata.h"
#include "../include/utils.h"
#include "../include/cuda_utils.cuh"
#include "../include/basis_to_gpu.cuh"
#include "../include/eval_rho_grad.cuh"

namespace py = pybind11;
using namespace py::literals;


TEST_CASE( "Test Derivative of Contractions Against gbasis", "[evaluate_contraction_derivatives]" ) {
  {  // Need this so that the python object doesn't outline the interpretor.
    // Get the IOdata object from the fchk file.
    std::string fchk_file = GENERATE(
        "./tests/data/atom_01_H_N01_M2_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/atom_he.fchk",
        "./tests/data/atom_be.fchk",
        "./tests/data/atom_be_f_pure_orbital.fchk",
        "./tests/data/atom_be_f_cartesian_orbital.fchk",
        "./tests/data/atom_kr.fchk",
        "./tests/data/atom_o.fchk",
        "./tests/data/atom_c_g_pure_orbital.fchk",
        "./tests/data/atom_mg.fchk",
        "./tests/data/E948_rwB97XD_def2SVP.fchk",
        "./tests/data/test.fchk",
        "./tests/data/test2.fchk",
        "./tests/data/atom_08_O_N08_M3_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/atom_08_O_N09_M2_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/h2o.fchk",
        "./tests/data/ch4.fchk",
        "./tests/data/qm9_000092_HF_cc-pVDZ.fchk",
        "./tests/data/qm9_000104_PBE1PBE_pcS-3.fchk"
    );
    std::cout << "FCHK FILE %s \n" << fchk_file << std::endl;
    chemtools::IOData iodata = chemtools::get_molecular_basis_from_fchk(fchk_file);

    // Gemerate random grid.
    int numb_pts = 10000;
    std::vector<double> points(3 * numb_pts);
    std::random_device rnd_device;
    std::mt19937  merseene_engine {rnd_device()};
    std::uniform_real_distribution<double> dist {-10, 10};
    auto gen = [&dist, &merseene_engine](){return dist(merseene_engine);};
    std::generate(points.begin(), points.end(), gen);

    // Calculate Gradient
    std::vector<double> gradient_result = chemtools::evaluate_contraction_derivatives(iodata, points.data(), numb_pts);

    // COnvert them (with copy) to python objects so that they can be transfered.
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
        py_result = chemtools::as_pyarray_from_vector(gradient_result);
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
        py_points = chemtools::as_pyarray_from_vector(points);

    const int nbasis = iodata.GetOrbitalBasis().numb_basis_functions();

    auto locals = py::dict("points"_a = py_points,
                           "true_result"_a = py_result,
                           "fchk_path"_a = fchk_file,
                           "numb_pts"_a = numb_pts,
                           "nbasis"_a = nbasis);
    py::exec(R"(
import numpy as np
from gbasis.evals.density import evaluate_deriv_basis, evaluate_basis
from iodata import load_one
from gbasis.wrappers import from_iodata

true_result = true_result.reshape((3, nbasis, numb_pts), order="C")  # row-major order
iodata = load_one(fchk_path)
basis = from_iodata(iodata)

points = points.reshape((numb_pts, 3), order="F")
points = np.array(points, dtype=np.float64)

# Derivative in X-coordinate
for i, deriv in enumerate([[1, 0, 0], [0, 1, 0], [0, 0, 1]]):

  derivative =  evaluate_deriv_basis(
              basis, points, np.array(deriv)
          )
  error = np.abs(derivative - true_result[i, :, :])
  print("Statistics", np.mean(derivative), np.max(derivative))

  print(deriv, np.max(error), np.mean(error), np.std(error))
  assert np.all(error < 1e-10), "Gradient on electron density on GPU doesn't match gbasis."

    )", py::globals(), locals);
  } // Need this so that the python object doesn't outline the interpretor when we close it up.
}



TEST_CASE( "Test Gradient of Electron Density Against gbasis", "[evaluate_electron_density_gradient]" ) {
  {  // Need this so that the python object doesn't outline the interpretor.
    // Get the IOdata object from the fchk file.
    std::string fchk_file = GENERATE(
        "./tests/data/atom_01_H_N01_M2_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/atom_he.fchk",
        "./tests/data/atom_be.fchk",
        "./tests/data/atom_be_f_pure_orbital.fchk",
        "./tests/data/atom_be_f_cartesian_orbital.fchk",
        "./tests/data/atom_kr.fchk",
        "./tests/data/atom_o.fchk",
        "./tests/data/atom_c_g_pure_orbital.fchk",
        "./tests/data/atom_mg.fchk",
        "./tests/data/E948_rwB97XD_def2SVP.fchk",
        "./tests/data/test.fchk",
        "./tests/data/test2.fchk",
        "./tests/data/atom_08_O_N08_M3_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/atom_08_O_N09_M2_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/h2o.fchk",
        "./tests/data/ch4.fchk",
        "./tests/data/qm9_000092_HF_cc-pVDZ.fchk",
        "./tests/data/qm9_000104_PBE1PBE_pcS-3.fchk",
        "./tests/data/DUTLAF10_0_q000_m01_k00_force_uwb97xd_def2svpd.fchk"
    );
    std::cout << "Gradient Test: FCHK file: %s \n" << fchk_file << std::endl;
    chemtools::IOData iodata = chemtools::get_molecular_basis_from_fchk(fchk_file);

    // Gemerate random grid.
    int numb_pts = 5000000;
    std::vector<double> points(3 * numb_pts);
    std::random_device rnd_device;
    std::mt19937  merseene_engine {rnd_device()};
    std::uniform_real_distribution<double> dist {-5, 5};
    auto gen = [&dist, &merseene_engine](){return dist(merseene_engine);};
    std::generate(points.begin(), points.end(), gen);

    // Calculate Gradient
    std::vector<double> gradient_result = chemtools::evaluate_electron_density_gradient(iodata, points.data(), numb_pts);

    // COnvert them (with copy) to python objects so that they can be transfered.
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
        py_result = chemtools::as_pyarray_from_vector(gradient_result);
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
        py_points = chemtools::as_pyarray_from_vector(points);

    auto locals = py::dict("points"_a = py_points,
                           "true_result"_a = py_result,
                           "fchk_path"_a = fchk_file,
                           "numb_pts"_a = numb_pts);
    py::exec(R"(
import numpy as np
from gbasis.evals.density import evaluate_density_gradient, evaluate_density, evaluate_deriv_basis
from iodata import load_one
from gbasis.wrappers import from_iodata

true_result = true_result.reshape((numb_pts, 3), order="C")  # Row-major order

iodata = load_one(fchk_path)
basis = from_iodata(iodata)
rdm = (iodata.mo.coeffs * iodata.mo.occs).dot(iodata.mo.coeffs.T)
points = points.reshape((numb_pts, 3), order="F")
points = np.array(points, dtype=np.float64)

indices_to_compute = np.unique(np.random.choice(np.arange(len(points)), size=10000))
true_result = true_result[indices_to_compute, :]
points = points[indices_to_compute, :]

gradient = evaluate_density_gradient(rdm, basis, points)
error = np.abs(gradient - true_result)
print("Max, Mean, STD , Min error ", np.max(error), np.mean(error), np.std(error), np.min(error))
assert np.all(error < 1e-10), "Gradient on electron density on GPU doesn't match gbasis."
    )", py::globals(), locals);
  } // Need this so that the python object doesn't outline the interpretor when we close it up.
}



TEST_CASE( "Test Gradient of Electron Density Against gbasis (Col Order)", "[evaluate_electron_density_gradient_col]" ) {
  {  // Need this so that the python object doesn't outline the interpretor.
    // Get the IOdata object from the fchk file.
    std::string fchk_file = GENERATE(
        "./tests/data/atom_01_H_N01_M2_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/atom_he.fchk",
        "./tests/data/atom_be.fchk",
        "./tests/data/atom_be_f_pure_orbital.fchk",
        "./tests/data/atom_be_f_cartesian_orbital.fchk",
        "./tests/data/atom_kr.fchk",
        "./tests/data/atom_o.fchk",
        "./tests/data/atom_c_g_pure_orbital.fchk",
        "./tests/data/atom_mg.fchk",
        "./tests/data/E948_rwB97XD_def2SVP.fchk",
        "./tests/data/test.fchk",
        "./tests/data/test2.fchk",
        "./tests/data/atom_08_O_N08_M3_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/atom_08_O_N09_M2_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/h2o.fchk",
        "./tests/data/ch4.fchk",
        "./tests/data/qm9_000092_HF_cc-pVDZ.fchk",
        "./tests/data/qm9_000104_PBE1PBE_pcS-3.fchk"
    );
    std::cout << "Gradient Test: FCHK FILE %s \n" << fchk_file << std::endl;
    chemtools::IOData iodata = chemtools::get_molecular_basis_from_fchk(fchk_file);

    // Gemerate random grid.
    int numb_pts = 700000;
    std::vector<double> points(3 * numb_pts);
    std::random_device rnd_device;
    std::mt19937  merseene_engine {rnd_device()};
    std::uniform_real_distribution<double> dist {-5, 5};
    auto gen = [&dist, &merseene_engine](){return dist(merseene_engine);};
    std::generate(points.begin(), points.end(), gen);

    // Calculate Gradient
    std::vector<double> gradient_result = chemtools::evaluate_electron_density_gradient(
        iodata, points.data(), numb_pts, false
    );

    // COnvert them (with copy) to python objects so that they can be transfered.
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
        py_result = chemtools::as_pyarray_from_vector(gradient_result);
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
        py_points = chemtools::as_pyarray_from_vector(points);

    auto locals = py::dict("points"_a = py_points,
                           "true_result"_a = py_result,
                           "fchk_path"_a = fchk_file,
                           "numb_pts"_a = numb_pts);
    py::exec(R"(
import numpy as np
from gbasis.evals.density import evaluate_density_gradient, evaluate_density
from iodata import load_one
from gbasis.wrappers import from_iodata

true_result = true_result.reshape((numb_pts, 3), order="F")  # Row-major order

iodata = load_one(fchk_path)
basis = from_iodata(iodata)
rdm = (iodata.mo.coeffs * iodata.mo.occs).dot(iodata.mo.coeffs.T)
points = points.reshape((numb_pts, 3), order="F")
points = np.array(points, dtype=np.float64)

indices_to_compute = np.random.choice(np.arange(len(points)), size=10000)
true_result = true_result[indices_to_compute, :]
points = points[indices_to_compute, :]

gradient = evaluate_density_gradient(rdm, basis, points)
error = np.abs(gradient - true_result)
print("Max, Mean, STD , Min error ", np.max(error), np.mean(error), np.std(error), np.min(error))
assert np.all(error < 1e-10), "Gradient on electron density on GPU doesn't match gbasis."
    )", py::globals(), locals);
  } // Need this so that the python object doesn't outline the interpretor when we close it up.
}