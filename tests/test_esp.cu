#include "hip/hip_runtime.h"
#include "catch.hpp"

#include <random>
#include <algorithm>
#include <iterator>

#include <pybind11/embed.h>
#include <pybind11/stl_bind.h>
#include <pybind11/numpy.h>

#include "../include/iodata.h"
#include "../include/utils.h"
#include "../include/cuda_utils.cuh"
#include "../include/basis_to_gpu.cuh"
#include "../include/eval_esp.cuh"

namespace py = pybind11;
using namespace py::literals;


TEST_CASE( "Test Electrostatic Potential", "[evaluate_electrostatic_potential]" ) {
  //py::initialize_interpreter();  // Open up the python interpretor for this test.
  {  // Need this so that the python object doesn't outline the interpretor.
    // Get the IOdata object from the fchk file.
    std::string fchk_file = GENERATE(
        "./tests/data/atom_01_H_N01_M2_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/atom_he.fchk",
        "./tests/data/atom_be.fchk",
        "./tests/data/atom_be_f_pure_orbital.fchk",
        "./tests/data/atom_be_f_cartesian_orbital.fchk",
        "./tests/data/atom_kr.fchk",
        "./tests/data/atom_o.fchk",
        // TODO: Support for g-type
        // "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/atom_c_g_pure_orbital.fchk",
        "./tests/data/atom_mg.fchk",
        "./tests/data/E948_rwB97XD_def2SVP.fchk",
        "./tests/data/test.fchk",
        "./tests/data/test2.fchk",
        "./tests/data/atom_08_O_N08_M3_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/atom_08_O_N09_M2_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/h2o.fchk",
        "./tests/data/ch4.fchk",
        "./tests/data/qm9_000092_HF_cc-pVDZ.fchk"
        // TODO: Support for g-type
        //"/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/qm9_000104_PBE1PBE_pcS-3.fchk"
    );
    std::cout << "FCHK FILE %s \n" << fchk_file << std::endl;
    chemtools::IOData iodata = chemtools::get_molecular_basis_from_fchk(fchk_file);

    // Gemerate random grid.
    int numb_pts = 1000;
    std::vector<double> points(3 * numb_pts);
    std::random_device rnd_device;
    std::mt19937  merseene_engine {rnd_device()};
    std::uniform_real_distribution<double> dist {-5, 5};
    auto gen = [&dist, &merseene_engine](){return dist(merseene_engine);};
    std::generate(points.begin(), points.end(), gen);

    // Calculate Gradient
    std::vector<double> esp_result = chemtools::compute_electrostatic_potential_over_points(
        iodata, points.data(), numb_pts
        );

    // COnvert them (with copy) to python objects so that they can be transfered.
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
        py_result = chemtools::as_pyarray_from_vector(esp_result);
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
        py_points = chemtools::as_pyarray_from_vector(points);

    const int nbasis = iodata.GetOrbitalBasis().numb_basis_functions();

    auto locals = py::dict("points"_a = py_points,
                           "true_result"_a = py_result,
                           "fchk_path"_a = fchk_file,
                           "numb_pts"_a = numb_pts,
                           "nbasis"_a = nbasis);
    py::exec(R"(
import numpy as np
from gbasis.evals.electrostatic_potential import electrostatic_potential
from iodata import load_one
from gbasis.wrappers import from_iodata

iodata = load_one(fchk_path)
basis = from_iodata(iodata)
points = points.reshape((numb_pts, 3), order="C")
points = np.array(points, dtype=np.float64)
rdm = (iodata.mo.coeffs * iodata.mo.occs).dot(iodata.mo.coeffs.T)

#from chemtools.wrappers import Molecule
#mol2 = Molecule.from_file(fchk_path)
#electro = mol2.compute_esp(points)
electro = electrostatic_potential(basis=basis, one_density_matrix=rdm, points=points, nuclear_coords=iodata.atcoords,
                                  nuclear_charges=iodata.atcorenums)
err = np.abs(electro - true_result)
print("Max error: ", np.max(err))
result = np.all(err < 1e-8)
assert result, "Electrostatic potential on GPU doesn't match gbasis."
    )", py::globals(), locals);
  } // Need this so that the python object doesn't outline the interpretor when we close it up.
  //py::finalize_interpreter(); // Close up the python interpretor for this test.
}

