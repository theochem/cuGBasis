#include "hip/hip_runtime.h"
#include "catch.hpp"

#include <random>
#include <algorithm>
#include <iterator>

#include <pybind11/embed.h>
#include <pybind11/stl_bind.h>
#include <pybind11/numpy.h>

#include "../include/iodata.h"
#include "../include/utils.h"
#include "../include/cuda_utils.cuh"
#include "../include/basis_to_gpu.cuh"
#include "../include/eval_lap.cuh"

namespace py = pybind11;
using namespace py::literals;


TEST_CASE( "Test Sum of Second Derivatives of Contractions Against gbasis",
           "[evaluate_sum_of_second_derivative_contractions_from_constant_memory]" ) {
  {  // Need this so that the python object doesn't outline the interpretor.
    // Get the IOdata object from the fchk file.
    std::string fchk_file = GENERATE(
        "./tests/data/atom_01_H_N01_M2_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/atom_he.fchk",
        "./tests/data/atom_be.fchk",
        "./tests/data/atom_be_f_pure_orbital.fchk",
        "./tests/data/atom_be_f_cartesian_orbital.fchk",
        "./tests/data/atom_kr.fchk",
        "./tests/data/atom_o.fchk",
        "./tests/data/atom_c_g_pure_orbital.fchk",
        "./tests/data/atom_mg.fchk",
        "./tests/data/E948_rwB97XD_def2SVP.fchk",
        "./tests/data/test.fchk",
        "./tests/data/test2.fchk",
        "./tests/data/atom_08_O_N08_M3_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/atom_08_O_N09_M2_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/h2o.fchk",
        "./tests/data/ch4.fchk",
        "./tests/data/qm9_000092_HF_cc-pVDZ.fchk",
        "./tests/data/qm9_000104_PBE1PBE_pcS-3.fchk"
    );
    std::cout << "Test Sum of Second Derivs: FCHK file %s \n" << fchk_file << std::endl;
    chemtools::IOData iodata = chemtools::get_molecular_basis_from_fchk(fchk_file);

    // Gemerate random grid.
    int numb_pts = 10000;
    std::vector<double> points(3 * numb_pts);
    std::random_device rnd_device;
    std::mt19937  merseene_engine {rnd_device()};
    std::uniform_real_distribution<double> dist {-5, 5};
    auto gen = [&dist, &merseene_engine](){return dist(merseene_engine);};
    std::generate(points.begin(), points.end(), gen);

    // Calculate Gradient
    std::vector<double> second_sum_deriv = chemtools::evaluate_sum_of_second_derivative_contractions(
        iodata, points.data(), numb_pts
        );

    // COnvert them (with copy) to python objects so that they can be transfered.
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
        py_result = chemtools::as_pyarray_from_vector(second_sum_deriv);
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
        py_points = chemtools::as_pyarray_from_vector(points);

    const int nbasis = iodata.GetOrbitalBasis().numb_basis_functions();

    auto locals = py::dict("points"_a = py_points,
                           "true_result"_a = py_result,
                           "fchk_path"_a = fchk_file,
                           "numb_pts"_a = numb_pts,
                           "nbasis"_a = nbasis);
    py::exec(R"(
import numpy as np
from gbasis.evals.density import evaluate_deriv_basis, evaluate_basis
from iodata import load_one
from gbasis.wrappers import from_iodata

true_result = true_result.reshape((nbasis, numb_pts), order="C")  # column-major order
iodata = load_one(fchk_path)
basis = from_iodata(iodata)

points = points.reshape((numb_pts, 3), order="F")
points = np.array(points, dtype=np.float64)

# Derivative in X-coordinate
output = np.zeros(true_result.shape)
for deriv in [[2, 0, 0], [0, 2, 0], [0, 0, 2]]:
  derivative =  evaluate_deriv_basis(
              basis, points, np.array(deriv)
          )
  output += derivative

error = np.abs(output - true_result)
print(np.max(error), np.mean(error), np.std(error))
assert np.all(error < 1e-10), "Gradient on electron density on GPU doesn't match gbasis."
    )", py::globals(), locals);
  } // Need this so that the python object doesn't outline the interpretor when we close it up.
}


TEST_CASE( "Test Laplacian of Electron Density Against gbasis", "[evaluate_laplacian]" ) {
  //py::initialize_interpreter();  // Open up the python interpretor for this test.
  {  // Need this so that the python object doesn't outline the interpretor.
    // Get the IOdata object from the fchk file.
    std::string fchk_file = GENERATE(
        "./tests/data/atom_01_H_N01_M2_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/atom_he.fchk",
        "./tests/data/atom_be.fchk",
        "./tests/data/atom_be_f_pure_orbital.fchk",
        "./tests/data/atom_be_f_cartesian_orbital.fchk",
        "./tests/data/atom_kr.fchk",
        "./tests/data/atom_o.fchk",
        "./tests/data/atom_c_g_pure_orbital.fchk",
        "./tests/data/atom_mg.fchk",
        "./tests/data/E948_rwB97XD_def2SVP.fchk",
        "./tests/data/test.fchk",
        "./tests/data/test2.fchk",
        "./tests/data/atom_08_O_N08_M3_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/atom_08_O_N09_M2_ub3lyp_ccpvtz_g09.fchk",
        "./tests/data/h2o.fchk",
        "./tests/data/ch4.fchk",
        "./tests/data/qm9_000092_HF_cc-pVDZ.fchk",
        "./tests/data/qm9_000104_PBE1PBE_pcS-3.fchk",
        "./tests/data/DUTLAF10_0_q000_m01_k00_force_uwb97xd_def2svpd.fchk"
    );
    std::cout << "Laplacian FCHK FILE %s \n" << fchk_file << std::endl;
    chemtools::IOData iodata = chemtools::get_molecular_basis_from_fchk(fchk_file);

    // Gemerate random grid.
    int numb_pts = 1000000;
    std::vector<double> points(3 * numb_pts);
    std::random_device rnd_device;
    std::mt19937  merseene_engine {rnd_device()};
    std::uniform_real_distribution<double> dist {-5, 5};
    auto gen = [&dist, &merseene_engine](){return dist(merseene_engine);};
    std::generate(points.begin(), points.end(), gen);

    // Calculate Gradient
    std::vector<double> laplacian_result = chemtools::evaluate_laplacian(
        iodata, points.data(), numb_pts
        );

    // COnvert them (with copy) to python objects so that they can be transfered.
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
        py_result = chemtools::as_pyarray_from_vector(laplacian_result);
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
        py_points = chemtools::as_pyarray_from_vector(points);

    const int nbasis = iodata.GetOrbitalBasis().numb_basis_functions();

    auto locals = py::dict("points"_a = py_points,
                           "true_result"_a = py_result,
                           "fchk_path"_a = fchk_file,
                           "numb_pts"_a = numb_pts,
                           "nbasis"_a = nbasis);
    py::exec(R"(
import numpy as np
from gbasis.evals.density import evaluate_density_laplacian, evaluate_deriv_reduced_density_matrix, evaluate_deriv_basis
from iodata import load_one
from gbasis.wrappers import from_iodata

iodata = load_one(fchk_path)
basis = from_iodata(iodata)
rdm = (iodata.mo.coeffs * iodata.mo.occs).dot(iodata.mo.coeffs.T)
points = points.reshape((numb_pts, 3), order="F")
points = np.array(points, dtype=np.float64)

indices_to_compute = np.unique(np.random.choice(np.arange(len(points)), size=10000))
true_result = true_result[indices_to_compute]
points = points[indices_to_compute, :]

laplacian = evaluate_density_laplacian(rdm, basis, points)
err = np.abs(laplacian - true_result)
result = np.all(err < 1e-8)
print("Mean, Max, STD Error ", np.mean(err), np.max(err), np.std(err))
assert result, "Laplacian of Electron Density on GPU doesn't match gbasis."
print("\n\n\n")
    )", py::globals(), locals);
  } // Need this so that the python object doesn't outline the interpretor when we close it up.
}
